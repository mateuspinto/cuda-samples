#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "common.cuh"

#define VECTOR_QNT 2000000
#define VECTOR_SIZE VECTOR_QNT*sizeof(int)

void initializeVector(int * vector)
{
    for(size_t i=0; i<VECTOR_QNT; i++)
    {
        vector[i] = i;
    }
}

void CPUVectorSum(int * a, int * b, int * c)
{
    for(size_t i=0; i<VECTOR_QNT; i++)
    {
        c[i] = a[i] + b[i];
    }
}

__global__ void GPUVectorSum(int * a, int * b, int * c) {
    int n = VECTOR_QNT;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < n; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}

int checkCorrect(int * vector)
{
    int errors = 0;

    for(size_t i=0; i<VECTOR_QNT; i++)
    {
        if(vector[i] != i*2)
        {
            errors++;
        }
    }

    return errors;
}

int main()
{
    int * a_cpu = (int*)malloc(VECTOR_SIZE);
    int * b_cpu = (int*)malloc(VECTOR_SIZE);
    int * c_cpu = (int*)malloc(VECTOR_SIZE);

    int * a_gpu = NULL;
    int * b_gpu = NULL;
    int * c_gpu = NULL;

    clock_t start, end;
    dim3 grid, block;

    initializeVector(a_cpu);
    initializeVector(b_cpu);

    // Allocating memory for GPU DMAs
    hipMalloc((int **)&a_gpu, VECTOR_SIZE);
    hipMalloc((int **)&b_gpu, VECTOR_SIZE);
    hipMalloc((int **)&c_gpu, VECTOR_SIZE);

    hipMemcpy(a_gpu, a_cpu, VECTOR_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, VECTOR_SIZE, hipMemcpyHostToDevice);

    start = clock();
    CPUVectorSum(a_cpu,b_cpu,c_cpu);
    end = clock();
    printf("Time=%f, Errors on CPU = %d\n", ((double) (end - start)) / CLOCKS_PER_SEC, checkCorrect(c_cpu));

    start = clock();
    GetGPUGridConfig(grid, block);
    GPUVectorSum<<<grid,block>>>(a_gpu,b_gpu,c_gpu);
    CheckGpuPanic();
    hipDeviceSynchronize();
    end = clock();

    hipMemcpy(c_cpu, c_gpu, VECTOR_SIZE, hipMemcpyDeviceToHost);
    printf("Time=%f, Errors on GPU = %d\n", ((double) (end - start)) / CLOCKS_PER_SEC, checkCorrect(c_cpu));

    free(a_cpu);
    free(b_cpu);
    free(c_cpu);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);

    return 0;
}